#include "hip/hip_runtime.h"
#include <stdio.h>
#include <mpi.h>

#include "functions.h"
#include "utils.h"

hipDeviceProp_t set_gpu_and_get_properties(int rank) {
    hipDeviceProp_t prop;
    int device_count, device;

    CUDA_CHECK(hipGetDeviceCount(&device_count), rank);

    if (device_count == 0) {
        fprintf(stderr, "Rank %d Error in get_gpu_properties: No CUDA-capable devices found.\n", rank);
        MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
    }
    
    device = 0;
    CUDA_CHECK(hipSetDevice(device), rank);
    CUDA_CHECK(hipGetDeviceProperties(&prop, device), rank);
    
    return prop;
}

int calculate_optimal_tile_width(hipDeviceProp_t prop, int rank) {
    int tile_width;

    int max_threads_per_block_sqrt = (int)sqrt((double)prop.maxThreadsPerBlock);

    for (tile_width = MIN(prop.warpSize, max_threads_per_block_sqrt); tile_width >= 4; tile_width--) {
        int threads_per_block = tile_width * tile_width;
        
        size_t required_shared_memory = 2 * threads_per_block * sizeof(double);

        if (required_shared_memory <= prop.sharedMemPerBlock) {
            return tile_width;
        }
    }

    fprintf(stderr, "Rank %d: Error: Unable to determine a suitable tile_width.\n", rank);
    MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
    return 0;
}


/**
 * Performs matrix multiplication C = A * B using a tiled approach
 * with shared memory to optimize global memory accesses.
 * 
 * Each thread block computes one tile of matrix C.
 * Within each block, threads cooperatively load the corresponding tiles
 * from matrices A and B into shared memory.
 * The multiplication is then performed using data from shared memory.
 * The process is iterated through "phases" to cover the entire K dimension.
 *
 * A Pointer to matrix A (M x K) in global memory.
 * B Pointer to matrix B (K x N) in global memory.
 * C Pointer to the resulting matrix C (M x N) in global memory.
 * M Number of rows in matrix A and matrix C.
 * N Number of columns in matrix B and matrix C.
 * K Number of columns in matrix A and number of rows in matrix B.
 */
__global__ void matrix_mul_kernel(double* A, double* B, double* C, int M, int N, int K) {

    extern __shared__ double shared_mem[];

    int tile_width = blockDim.x;

    double* s_A = (double*) shared_mem;
    double* s_B = (double*) shared_mem + tile_width * tile_width;

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * tile_width + ty;
    int col = bx * tile_width + tx;

    int phase;
    double c_value = 0;
    for(phase = 0; phase < ceil(K/(float)tile_width); ++phase) {
        if((row < M) && (phase * tile_width + tx) < K)
            s_A[ty * tile_width + tx] = A[row * K + phase * tile_width + tx];
        else
            s_A[ty * tile_width + tx] = 0.0;

        if((phase * tile_width + ty) < K && (col < N))
            s_B[ty * tile_width + tx] = B[(phase * tile_width + ty) * N + col];
        else
            s_B[ty * tile_width + tx] = 0.0;
        
        __syncthreads();

        for (int k = 0; k < tile_width; ++k) {
            c_value += s_A[ty * tile_width + k] * s_B[k * tile_width + tx];
        }
        __syncthreads();
    }

    if ((row < M) && (col < N))
        C[row * N + col] = c_value;
}