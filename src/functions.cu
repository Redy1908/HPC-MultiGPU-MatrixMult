#include "hip/hip_runtime.h"
#include <common_functions.h>
#include <math.h>
#include <stdlib.h>

#include "functions.cuh"
#include "utils.h"

hipDeviceProp_t set_gpu_and_get_properties(int rank) {
  hipDeviceProp_t prop;
  int device_count, device;

  CUDA_CHECK(hipGetDeviceCount(&device_count), rank);

  if (device_count == 0) {
    fprintf(
        stderr,
        "Rank %d Error in get_gpu_properties: No CUDA-capable devices found.\n",
        rank);
    MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
  }

  device = 0;
  CUDA_CHECK(hipSetDevice(device), rank);
  CUDA_CHECK(hipGetDeviceProperties(&prop, device), rank);

  return prop;
}

void check_threads_per_block(hipDeviceProp_t prop, int tile_width, int rank) {
  if (rank == 0) {
    int threads_per_block = tile_width * tile_width;

    if (threads_per_block > prop.maxThreadsPerBlock) {
      fprintf(stderr,
              "Rank %d: Error: Threads per block (%d) exceeds GPU max threads per block (%d).\n",
              rank, threads_per_block, prop.maxThreadsPerBlock);
      MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
    }
  }
}

void check_shared_memory_usage(hipDeviceProp_t prop, int tile_width, int rank) {
  if (rank == 0) {
    int threads_per_block = tile_width * tile_width;

    int required_shared_memory_size = 2 * threads_per_block * sizeof(double);

    if (required_shared_memory_size > prop.sharedMemPerBlock) {
      fprintf(stderr,
              "Rank %d: Warning: Required shared memory size (%d bytes) exceeds "
              "available shared memory (%zu bytes) per block. Performance will be affected.\n",
              rank, required_shared_memory_size, prop.sharedMemPerBlock);
    }
  }
}

int SUMMA(MPI_Comm grid_comm, double *A, double *B, double *C, uint m, uint k, uint n, dim3 grid_size, dim3 block_size) {
  int dims[2], periods[2], coords[2];
  MPI_Cart_get(grid_comm, 2, dims, periods, coords);

  int K2 = find_lcm(dims[0], dims[1]);

  uint a_block_height = m / dims[0];
  uint a_block_width = k / K2;

  uint b_block_height = a_block_width;
  uint b_block_width = n / dims[1];

  double *a_block = (double *)malloc(a_block_height * a_block_width * sizeof(double));
  double *b_block = (double *)malloc(b_block_height * b_block_width * sizeof(double));

  double *A_dev, *B_dev, *C_dev;
  hipMalloc(&A_dev, a_block_height * a_block_width * sizeof(double));
  hipMalloc(&B_dev, b_block_height * b_block_width * sizeof(double));
  hipMalloc(&C_dev, a_block_height * b_block_width * sizeof(double));

  if (a_block == NULL || b_block == NULL || A_dev == NULL || B_dev == NULL || C_dev == NULL) {
    hipFree(A_dev);
    hipFree(B_dev);
    hipFree(C_dev);

    free(a_block);
    free(b_block);

    return 1;
  }

  /* create communicators along rows and columns */
  int remain_dims_row[2] = {0, 1};
  int remain_dims_col[2] = {1, 0};
  MPI_Comm row_comm, col_comm;
  MPI_Cart_sub(grid_comm, remain_dims_row, &row_comm);
  MPI_Cart_sub(grid_comm, remain_dims_col, &col_comm);

  double *A_start = A;
  double *B_start = B;

  for (uint i = 0; i < K2; i++) {
    uint r = i % dims[0];
    uint c = i % dims[1];

    if (coords[1] == c) {
      for (uint j = 0; j < a_block_height; j++)
        memcpy(a_block + j * a_block_width, A_start + j * i, a_block_width * sizeof(double));

      A_start += a_block_width;
    }

    if (coords[0] == r) {
      for (uint j = 0; j < b_block_height; j++)
        memcpy(b_block + j * b_block_width, A_start + j * i, b_block_width * sizeof(double));

      B_start += b_block_width * n;
    }

    MPI_Bcast(a_block, a_block_height * a_block_width, MPI_DOUBLE, c, row_comm);
    MPI_Bcast(b_block, b_block_height * b_block_width, MPI_DOUBLE, r, col_comm);

    /* compute submatrix multiplication on the GPU */
    uint shared_mem_size = 2 * block_size.x * block_size.y * sizeof(double);
    hipMemcpy(A_dev, a_block, a_block_height * a_block_width * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(B_dev, b_block, b_block_height * b_block_width * sizeof(double), hipMemcpyHostToDevice);
    matrix_mul_kernel<<<grid_size, block_size, shared_mem_size>>>(A_dev, B_dev, C_dev, a_block_height, b_block_width, a_block_width);
    hipDeviceSynchronize(); /* TODO: is this needed? */
  }

  /* copy the final result from the GPU to the CPU */
  hipDeviceSynchronize();
  hipMemcpy(C, C_dev, a_block_height * b_block_width * sizeof(double), hipMemcpyDeviceToHost);

  /* cleanup */
  MPI_Comm_free(&row_comm);
  MPI_Comm_free(&col_comm);

  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(C_dev);

  free(a_block);
  free(b_block);

  return 0;
}

/**
 * Performs matrix multiplication C = A * B using a tiled approach
 * with shared memory to optimize global memory accesses.
 *
 * Each thread block computes one tile of matrix C.
 * Within each block, threads cooperatively load the corresponding tiles
 * from matrices A and B into shared memory.
 * The multiplication is then performed using data from shared memory.
 * The process is iterated through "phases" to cover the entire K dimension.
 *
 * A Pointer to matrix A (M x K) in global memory.
 * B Pointer to matrix B (K x N) in global memory.
 * C Pointer to the resulting matrix C (M x N) in global memory.
 * M Number of rows in matrix A and matrix C.
 * N Number of columns in matrix B and matrix C.
 * K Number of columns in matrix A and number of rows in matrix B.
 */
__global__ void matrix_mul_kernel(double *A, double *B, double *C, int M, int N, int K) {
  extern __shared__ double shared_mem[];

  int tile_width = blockDim.x;

  double *s_A = (double *)shared_mem;
  double *s_B = (double *)shared_mem + tile_width * tile_width;

  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = by * tile_width + ty;
  int col = bx * tile_width + tx;

  double c_value = 0.0;
  for (int phase = 0; phase < ceil(K / (float)tile_width); ++phase) {
    if ((row < M) && (phase * tile_width + tx) < K)
      s_A[ty * tile_width + tx] = A[row * K + phase * tile_width + tx];
    else
      s_A[ty * tile_width + tx] = 0.0;

    if ((phase * tile_width + ty) < K && (col < N))
      s_B[ty * tile_width + tx] = B[(phase * tile_width + ty) * N + col];
    else
      s_B[ty * tile_width + tx] = 0.0;

    __syncthreads();

    for (int k = 0; k < tile_width; ++k) {
      c_value += s_A[ty * tile_width + k] * s_B[k * tile_width + tx];
    }
    __syncthreads();
  }

  if ((row < M) && (col < N))
    C[row * N + col] += c_value;
}