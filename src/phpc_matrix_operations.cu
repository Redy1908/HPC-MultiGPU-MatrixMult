#include "hip/hip_runtime.h"
#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>

#include "phpc_matrix_operations.cuh"
#include "utils.cuh"

__global__ void gemm_kernel(double *A, double *B, double *C, int M, int N, int K) {
  extern __shared__ double shared_mem[];

  int tile_width = blockDim.x;

  double *s_A = (double *)shared_mem;
  double *s_B = (double *)shared_mem + tile_width * tile_width;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int num_tiles_along_N = (int)ceil((double)N / tile_width);
  int num_tiles_along_M = (int)ceil((double)M / tile_width);
  int total_output_tiles = num_tiles_along_M * num_tiles_along_N;

  int block_id_1d = blockIdx.y * gridDim.x + blockIdx.x;
  int total_launched_blocks = gridDim.x * gridDim.y;

  // Grid-stride loop: ogni blocco itera sulle tile di C che gli sono assegnate
  for (int current_tile_1d_idx = block_id_1d; current_tile_1d_idx < total_output_tiles; current_tile_1d_idx += total_launched_blocks) {
    int by_tile = current_tile_1d_idx / num_tiles_along_N;
    int bx_tile = current_tile_1d_idx % num_tiles_along_N;

    int C_tile_row_base = by_tile * tile_width;
    int C_tile_col_base = bx_tile * tile_width;

    int global_row_C = C_tile_row_base + ty;
    int global_col_C = C_tile_col_base + tx;

    double c_value = 0.0;

    int num_phases = (int)ceil((double)K / tile_width);
    for (int phase = 0; phase < num_phases; ++phase) {
      if ((global_row_C < M) && (phase * tile_width + tx) < K)
        s_A[ty * tile_width + tx] = A[global_row_C * K + phase * tile_width + tx];
      else
        s_A[ty * tile_width + tx] = 0.0;

      if ((phase * tile_width + ty) < K && (global_col_C < N))
        s_B[ty * tile_width + tx] = B[(phase * tile_width + ty) * N + global_col_C];
      else
        s_B[ty * tile_width + tx] = 0.0;

      __syncthreads();

      for (int k_tile = 0; k_tile < tile_width; ++k_tile) {
        if (phase * tile_width + k_tile < K) {
          c_value += s_A[ty * tile_width + k_tile] * s_B[k_tile * tile_width + tx];
        }
      }
      __syncthreads();
    }

    if ((global_row_C < M) && (global_col_C < N))
      C[global_row_C * N + global_col_C] += c_value;
  }
}

int phpc_gemm_cuda(const double *a, int lda, const double *b, int ldb, double *c, int ldc, int width, int gpu_count, int grid_width, int grid_height, int block_width) {
  int device_count;
  hipGetDeviceCount(&device_count);

  assert(width > 0);
  assert(gpu_count > 0 && gpu_count <= device_count);
  assert(grid_width > 0 && grid_height > 0);
  assert(block_width * block_width <= 1024);
  assert(width % gpu_count == 0);

  /**
   * Matrix A: each gpu has a "row"
   *  ________________________
   * |         GPU 0         |
   * -------------------------
   * |          ...          |
   * -------------------------
   * |         GPU N         |
   * -------------------------
   *
   * Matrix B: each gpu has a "column"
   *  ________________________
   * |       |       |       |
   * |       |       |       |
   * | GPU 0 |  ...  | GPU N |
   * |       |       |       |
   * |       |       |       |
   * -------------------------
   */

  int m = width / gpu_count;
  int k = width;

  double **dev_buffers_a = (double **)malloc(gpu_count * sizeof(double *));
  double **dev_buffers_b = (double **)malloc(gpu_count * sizeof(double *));
  double **dev_buffers_c = (double **)malloc(gpu_count * sizeof(double *));
  hipStream_t *streams = (hipStream_t *)malloc(gpu_count * sizeof(hipStream_t));

  assert(dev_buffers_a != NULL);
  assert(dev_buffers_b != NULL);
  assert(dev_buffers_c != NULL);
  assert(streams != NULL);

  for (int i = 0; i < gpu_count; i++) {
    hipSetDevice(i);
    hipMalloc(&(dev_buffers_a[i]), m * k * sizeof(double));
    hipMalloc(&(dev_buffers_b[i]), m * k * sizeof(double));
    hipMalloc(&(dev_buffers_c[i]), m * k * sizeof(double));
    hipStreamCreate(&(streams[i]));

    /* copy column of B*/
    hipMemcpy2DAsync(dev_buffers_b[i], m * sizeof(double), b + m * i, ldb * sizeof(double), m * sizeof(double), width, hipMemcpyHostToDevice, streams[i]);

    /* copy column of C */
    hipMemcpy2DAsync(dev_buffers_c[i], m * sizeof(double), c + m * i, ldc * sizeof(double), m * sizeof(double), width, hipMemcpyHostToDevice, streams[i]);
  }

  dim3 grid_size(grid_width, grid_height, 1);
  dim3 block_size(block_width, block_width, 1);
  int shared_memory_size = 2 * block_width * block_width * sizeof(double);

  for (int i = 0; i < gpu_count; i++) {
    for (size_t j = 0; j < gpu_count; j++) {
      hipSetDevice(j);

      /* copy row j % gpu_count of A */
      int row = (j + i) % gpu_count;
      hipMemcpyAsync(dev_buffers_a[j], a + m * k * row, m * k * sizeof(double), hipMemcpyHostToDevice, streams[j]);

      int dev_offset_c = m * m * row;
      gemm_kernel<<<grid_size, block_size, shared_memory_size, streams[j]>>>(dev_buffers_a[j], dev_buffers_b[j], dev_buffers_c[j] + dev_offset_c, m, m, k);
    }
  }

  /**
   * Gather results: each GPU has a column of the resulting matrix C
   *  ________________________
   * |       |       |       |
   * |       |       |       |
   * | GPU 0 |  ...  | GPU i |
   * |       |       |       |
   * |       |       |       |
   * -------------------------
   */
  for (int i = 0; i < gpu_count; i++) {
    hipSetDevice(i);
    hipMemcpy2D(c + m * i, width * sizeof(double), dev_buffers_c[i], m * sizeof(double), m * sizeof(double), width, hipMemcpyDeviceToHost);

    hipFree(&(dev_buffers_c[i]));
    hipFree(&(dev_buffers_b[i]));
    hipFree(&(dev_buffers_a[i]));
    hipStreamDestroy(streams[i]);
  }

  free(streams);
  free(dev_buffers_c);
  free(dev_buffers_b);
  free(dev_buffers_a);

  return 0;
}

void phpc_gemm_summa_cuda(MPI_Comm grid_comm, const double *A, const double *B, double *C, int lda, int ldb, int ldc, int matrices_width, int gpu_count, int grid_width, int grid_height, int block_width) {
  assert(lda > 0);
  assert(ldb > 0);
  assert(ldc > 0);

  MPI_Comm row_comm, col_comm;
  int dims[2], periods[2], coords[2];
  int remain_dims_row[2] = {0, 1};
  int remain_dims_col[2] = {1, 0};
  MPI_Cart_get(grid_comm, 2, dims, periods, coords);
  MPI_Cart_sub(grid_comm, remain_dims_row, &row_comm);
  MPI_Cart_sub(grid_comm, remain_dims_col, &col_comm);

  int lcm = find_lcm(dims[0], dims[1]);
  int local_A_rows = matrices_width / dims[0];
  int panel_K_dim = matrices_width / lcm;
  int local_B_cols = matrices_width / dims[1];

  assert(matrices_width % dims[0] == 0);
  assert(matrices_width % lcm == 0);

  /* compute optimal size */
  if (grid_width == 0 && grid_height == 0) {
    grid_width = local_B_cols / block_width + (local_B_cols % block_width > 0);
    grid_height = local_A_rows / block_width + (local_A_rows % block_width > 0);
  }

  double *buffer_a = (double *)malloc(local_A_rows * panel_K_dim * sizeof(double));
  double *buffer_b = (double *)malloc(panel_K_dim * local_B_cols * sizeof(double));

  assert(buffer_a != NULL);
  assert(buffer_b != NULL);

  for (int k = 0; k < lcm; k++) {
    int sender_column = k % dims[1];
    int sender_row = k % dims[0];

    const double *block_a, *block_b;

    if (coords[1] == sender_column) {
      block_a = A;                     /* we are sending the block */
      A += local_A_rows * panel_K_dim; /* we may have to send again in the future, skip the pointer to the start of the other block */
    } else {
      block_a = buffer_a; /* we are receiving, prepare the buffer */
    }

    if (coords[0] == sender_row) {
      block_b = B;                     /* we are sending the block */
      B += panel_K_dim * local_B_cols; /* we may have to send again in the future, skip the pointer to the start of the other block */
    } else {
      block_b = buffer_b; /* we are receiving, prepare the buffer */
    }

    MPI_Bcast((void *)block_a, local_A_rows * panel_K_dim, MPI_DOUBLE, sender_column, row_comm);
    MPI_Bcast((void *)block_b, panel_K_dim * local_B_cols, MPI_DOUBLE, sender_row, col_comm);

    phpc_gemm_cuda(block_a, lda, block_b, ldb, C, ldc, matrices_width, gpu_count, grid_width, grid_height, block_width);
  }

  // TODO: gather matrices

  free(buffer_b);
  free(buffer_a);

  MPI_Comm_free(&row_comm);
  MPI_Comm_free(&col_comm);
}
