#include "hip/hip_runtime.h"
#include <assert.h>
#include <cublasXt.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>

#include "phpc_matrix_operations.cuh"
#include "utils.cuh"

typedef void (*gemm_t)(const double *a, int lda, const double *b, int ldb, double *c, int ldc, int m, int k, int n, int gpu_count, int grid_width, int grid_height, int block_width);

void phpc_gemm_iterative(const double *A, const double *B, double *C, int N) {
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      for (int k = 0; k < N; ++k) {
        C[i * N + k] += A[i * N + j] * B[j * N + k];
      }
    }
  }
}

__global__ void gemm_kernel(double *A, double *B, double *C, int M, int N, int K) {
  extern __shared__ double shared_mem[];

  int tile_width = blockDim.x;

  double *s_A = (double *)shared_mem;
  double *s_B = (double *)shared_mem + tile_width * tile_width;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int num_tiles_along_N = (int)ceil((double)N / tile_width);
  int num_tiles_along_M = (int)ceil((double)M / tile_width);
  int total_output_tiles = num_tiles_along_M * num_tiles_along_N;

  int block_id_1d = blockIdx.y * gridDim.x + blockIdx.x;
  int total_launched_blocks = gridDim.x * gridDim.y;

  // Grid-stride loop: each block processes multiple tiles
  for (int current_tile_1d_idx = block_id_1d; current_tile_1d_idx < total_output_tiles; current_tile_1d_idx += total_launched_blocks) {
    int by_tile = current_tile_1d_idx / num_tiles_along_N;
    int bx_tile = current_tile_1d_idx % num_tiles_along_N;

    int global_row_C = by_tile * tile_width + ty;
    int global_col_C = bx_tile * tile_width + tx;

    double c_value = 0.0;

    int phases = K / tile_width + (K / tile_width != 0);
    for (int phase = 0; phase < phases; ++phase) {
      if ((global_row_C < M) && (phase * tile_width + tx) < K)
        s_A[ty * tile_width + tx] = A[global_row_C * K + phase * tile_width + tx];
      else
        s_A[ty * tile_width + tx] = 0.0;

      if ((phase * tile_width + ty) < K && (global_col_C < N))
        s_B[ty * tile_width + tx] = B[(phase * tile_width + ty) * N + global_col_C];
      else
        s_B[ty * tile_width + tx] = 0.0;

      __syncthreads();

      for (int k_tile = 0; k_tile < tile_width; ++k_tile)
        c_value += s_A[ty * tile_width + k_tile] * s_B[k_tile * tile_width + tx];

      __syncthreads();
    }

    if ((global_row_C < M) && (global_col_C < N))
      C[global_row_C * N + global_col_C] += c_value;
  }
}

void phpc_gemm_cublas(const double *a, int lda, const double *b, int ldb, double *c, int ldc, int m, int k, int n, int gpu_count, int grid_width, int grid_height, int block_width) {
  int devices[32]; /* checking for 32 devices on a single machine is more than enough */
  cublasXtHandle_t handle;
  double alpha = 1, beta = 1;

  for (size_t i = 0; i < gpu_count; i++)
    devices[i] = i;

  cublasXtCreate(&handle);
  cublasXtDeviceSelect(handle, gpu_count, devices);

  /* note: some subtle math magic to make it work since cublas expects column-major matrices https://stackoverflow.com/a/56064726/17731255 */
  cublasXtDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, b, ldb, a, lda, &beta, c, ldc);
  cublasXtDestroy(handle);
}

void phpc_gemm_cuda(const double *a, int lda, const double *b, int ldb, double *c, int ldc, int m, int k, int n, int gpu_count, int grid_width, int grid_height, int block_width) {
  int max_shared_memory_per_block;
  hipDeviceGetAttribute(&max_shared_memory_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

  int required_shared_memory = 2 * block_width * block_width * sizeof(double);

  if (required_shared_memory > max_shared_memory_per_block)
    printf("Warning: required shared memory exceeds the GPU block limit. This will impact performance.\n");

  /**
   * Matrix A: each gpu copies the entire matrix
   *
   * Matrix B: each gpu has a "column"
   *  ________________________
   * |       |       |       |
   * |       |       |       |
   * | GPU 0 |  ...  | GPU N |
   * |       |       |       |
   * |       |       |       |
   * -------------------------
   *
   * Matrix C: each gpu has a resulting "column"
   *  ________________________
   * |       |       |       |
   * |       |       |       |
   * | GPU 0 |  ...  | GPU i |
   * |       |       |       |
   * |       |       |       |
   * -------------------------
   */

  dim3 grid_size(grid_width, grid_height, 1);
  dim3 block_size(block_width, block_width, 1);

  double **dev_buffers_a = (double **)malloc(gpu_count * sizeof(double *));
  double **dev_buffers_b = (double **)malloc(gpu_count * sizeof(double *));
  double **dev_buffers_c = (double **)malloc(gpu_count * sizeof(double *));
  hipStream_t *streams = (hipStream_t *)malloc(gpu_count * sizeof(hipStream_t));

  for (int gpu = 0; gpu < gpu_count; gpu++) {
    int dev_n = n / gpu_count + (gpu < n % gpu_count);

    hipSetDevice(gpu);
    hipStreamCreate(&(streams[gpu]));

    hipMallocAsync(&(dev_buffers_a[gpu]), m * k * sizeof(double), streams[gpu]);
    hipMallocAsync(&(dev_buffers_b[gpu]), k * dev_n * sizeof(double), streams[gpu]);
    hipMallocAsync(&(dev_buffers_c[gpu]), m * dev_n * sizeof(double), streams[gpu]);

    /* copy from host to device */
    hipMemcpy2DAsync(dev_buffers_a[gpu], k * sizeof(double), a, lda * sizeof(double), k * sizeof(double), m, hipMemcpyHostToDevice, streams[gpu]);
    hipMemcpy2DAsync(dev_buffers_b[gpu], dev_n * sizeof(double), b, ldb * sizeof(double), dev_n * sizeof(double), k, hipMemcpyHostToDevice, streams[gpu]);
    hipMemcpy2DAsync(dev_buffers_c[gpu], dev_n * sizeof(double), c, ldc * sizeof(double), dev_n * sizeof(double), m, hipMemcpyHostToDevice, streams[gpu]);

    /* perform computation */
    gemm_kernel<<<grid_size, block_size, required_shared_memory, streams[gpu]>>>(dev_buffers_a[gpu], dev_buffers_b[gpu], dev_buffers_c[gpu], m, dev_n, k);

    /* copy result from device to host */
    hipMemcpy2DAsync(c, ldc * sizeof(double), dev_buffers_c[gpu], dev_n * sizeof(double), dev_n * sizeof(double), m, hipMemcpyDeviceToHost, streams[gpu]);

    hipFreeAsync(dev_buffers_c[gpu], streams[gpu]);
    hipFreeAsync(dev_buffers_b[gpu], streams[gpu]);
    hipFreeAsync(dev_buffers_a[gpu], streams[gpu]);

    b += dev_n;
    c += dev_n;
  }

  for (int gpu = 0; gpu < gpu_count; gpu++) {
    hipSetDevice(gpu);
    hipStreamSynchronize(streams[gpu]);
    hipStreamDestroy(streams[gpu]);
  }

  free(streams);
  free(dev_buffers_c);
  free(dev_buffers_b);
  free(dev_buffers_a);
}

int phpc_gemm_summa(gemm_t f, MPI_Comm grid_comm, const double *A, const double *B, double *C, int N, int gpu_count, int grid_width, int grid_height, int block_width) {
  /* get MPI properties */
  int rank, size, dims[2], periods[2], coords[2];
  int remain_dims_row[2] = {0, 1};
  int remain_dims_col[2] = {1, 0};
  MPI_Comm row_comm, col_comm;
  MPI_Comm_rank(grid_comm, &rank);
  MPI_Comm_size(grid_comm, &size);
  MPI_Cart_get(grid_comm, 2, dims, periods, coords);
  MPI_Cart_sub(grid_comm, remain_dims_row, &row_comm);
  MPI_Cart_sub(grid_comm, remain_dims_col, &col_comm);

  int lcm = find_lcm(dims[0], dims[1]);
  int local_A_rows = N / dims[0];
  int panel_K_dim = N / lcm;
  int local_B_cols = N / dims[1];

  /* shift the start of the matrices to the first block actually corresponding to the process */
  A += coords[0] * N * local_A_rows + coords[1] * panel_K_dim;
  B += coords[0] * N * panel_K_dim + coords[1] * local_B_cols;
  double *offset_c = C + coords[0] * N * local_A_rows + coords[1] * local_B_cols;

  /* prepare buffers to receive blocks from other processes */
  double *buffer_a = (double *)malloc(local_A_rows * panel_K_dim * sizeof(double));
  double *buffer_b = (double *)malloc(panel_K_dim * local_B_cols * sizeof(double));

  /* create derived datatypes to exchange the blocks across the network */
  /* this is due the fact the blocks a process must handle are a portion than the actual dimension of the matrices */
  /* rows of each block are not contiguous in memory */
  MPI_Datatype block_a_type, block_b_type, block_c_type;
  MPI_Type_vector(local_A_rows, panel_K_dim, N, MPI_DOUBLE, &block_a_type);
  MPI_Type_vector(panel_K_dim, local_B_cols, N, MPI_DOUBLE, &block_b_type);
  MPI_Type_vector(local_A_rows, local_B_cols, N, MPI_DOUBLE, &block_c_type);
  MPI_Type_commit(&block_a_type);
  MPI_Type_commit(&block_b_type);
  MPI_Type_commit(&block_c_type);

  for (int k = 0; k < lcm; k++) {
    int sender_column = k % dims[1];
    int sender_row = k % dims[0];

    int block_lda = panel_K_dim;     /* the leading dimension of the block A to use in this step */
    int block_ldb = local_B_cols;    /* the leading dimension of the block B to use in this step */
    const double *block_a, *block_b; /* pointers to the start of the blocks to use in this step */

    if (coords[1] == sender_column) {
      block_a = A;                                                          /* we are sending the block */
      block_lda = N;                                                        /* set the leading dimension to the one of the original matrix */
      A += dims[1] * panel_K_dim;                                           /* we may have to send again in the future, skip the pointer to the start of the other block assigned to the process */
      MPI_Bcast((void *)block_a, 1, block_a_type, sender_column, row_comm); /* send the block as a composite data type, so that multiple lines are received as contiguous */
    } else {
      block_a = buffer_a;                                                                          /* we are receiving, prepare the buffer */
      MPI_Bcast((void *)block_a, local_A_rows * panel_K_dim, MPI_DOUBLE, sender_column, row_comm); /* receive the block as a contiguous array */
    }

    if (coords[0] == sender_row) {
      block_b = B;                                                       /* we are sending the block */
      block_ldb = N;                                                     /* set the leading dimension to the one of the original matrix */
      B += dims[0] * panel_K_dim * N;                                    /* we may have to send again in the future, skip the pointer to the start of the other block assigned to the process */
      MPI_Bcast((void *)block_b, 1, block_b_type, sender_row, col_comm); /* send the block as a composite data type, so that multiple lines are received as contiguous */
    } else {
      block_b = buffer_b;                                                                       /* we are receiving, prepare the buffer */
      MPI_Bcast((void *)block_b, panel_K_dim * local_B_cols, MPI_DOUBLE, sender_row, col_comm); /* receive the block as a contiguous array */
    }

    /* compute product of the blocks */
    f(block_a, block_lda, block_b, block_ldb, offset_c, N, local_A_rows, panel_K_dim, local_B_cols, gpu_count, grid_width, grid_height, block_width);
  }

  if (rank == 0) {
    /* process 0 receives from all other processes */
    for (int i = 1; i < size; i++) {
      int sender_coords[2];
      MPI_Cart_coords(grid_comm, i, 2, sender_coords);

      double *c_dest = C + N * sender_coords[0] * local_A_rows + sender_coords[1] * local_B_cols;
      MPI_Recv(c_dest, 1, block_c_type, i, 0, grid_comm, MPI_STATUS_IGNORE);
    }
  } else {
    /* all other processes send their results to process 0 */
    double *c_start = C + N * coords[0] * local_A_rows + coords[1] * local_B_cols;
    MPI_Send(c_start, 1, block_c_type, 0, 0, grid_comm);
  }

  MPI_Type_free(&block_c_type);
  MPI_Type_free(&block_b_type);
  MPI_Type_free(&block_a_type);

  free(buffer_b);
  free(buffer_a);
  MPI_Comm_free(&row_comm);
  MPI_Comm_free(&col_comm);

  return 0;
}

void phpc_gemm_summa_cuda(MPI_Comm grid_comm, const double *A, const double *B, double *C, int N, int gpu_count, int grid_width, int grid_height, int block_width) {
  phpc_gemm_summa(phpc_gemm_cuda, grid_comm, A, B, C, N, gpu_count, grid_width, grid_height, block_width);
}

void phpc_gemm_summa_cublas(MPI_Comm grid_comm, const double *A, const double *B, double *C, int N, int gpu_count) {
  phpc_gemm_summa(phpc_gemm_cublas, grid_comm, A, B, C, N, gpu_count, 0, 0, 0);
}
